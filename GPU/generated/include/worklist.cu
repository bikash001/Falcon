
#include <hip/hip_runtime.h>
__global__ void __worklist_initialize__(int size, int count, int *arr) {
	int id = threadIdx.x * blockDim.y + threadIdx.y;
	if(id < size) {
		int n = max((id+1)*count, size);
		for(int i=id*count; i<n; ++i) {
			arr[i] = i;
		}
	}
}


class Worklist
{
private:
	int *in, *out;
	int inSize;
	int outSize;
	int arrSize;

	__host__ bool allocate(int sz) {
		arrSize = sz;
		if(hipMalloc((void**)&in, sizeof(int)*sz) != hipSuccess) {
			return false;
		}
		return hipMalloc((void**)&out, sizeof(int)*sz) == hipSuccess;
	}

	__host__ void initialize(int i) {
		hipMemcpyToSymbol(HIP_SYMBOL(in), &i, sizeof(int), 0, hipMemcpyHostToDevice);
		inSize = 1;
		outSize = 0;
	}

public:
	__host__ Worklist(int npoints, int nedges, int start=0) {
		if(!allocate(nedges)) {
			fprintf(stderr, "ERROR: %s\n", "Memory allocation error @Worklist");
		}
		initialize(start);
		inSize = 1;
	}

	__device__ void push(int val) {
		int i = atomicAdd(&outSize, 1);
		out[i] = val;
	}

	__device__ int get(int pos) {
		return in[pos];
	}

	__device__ int get() {
		int i = atomicAdd(&inSize, 1);
		return in[i];
	}

	__host__ __device__ bool empty() {
		return inSize == 0;
	}

	__host__ void swap() {
		inSize = outSize;
		outSize = 0;
		int *temp = in;
		in = out;
		out = temp;
	}

	__host__ __device__ int size() {
		return inSize;
	}

	~Worklist() {
	#ifndef __CUDA_ARCH__
		hipFree(in);
		hipFree(out);
	#endif
	}
};