#include "hip/hip_runtime.h"
__global__ void __worklist_initialize__(int size, int count, int *arr) {
	int id = threadIdx.x * blockDim.y + threadIdx.y;
	if(id < size) {
		int n = max((id+1)*count, size);
		for(int i=id*count; i<n; ++i) {
			arr[i] = i;
		}
	}
}


class Worklist
{
private:
	int *in, *out;
	int inSize;
	int outSize;
	int arrSize;

	__host__ bool allocate(int sz) {
		arrSize = sz;
		if(hipMalloc((void**)&in, sizeof(int)*sz) != hipSuccess) {
			return false;
		}
		return hipMalloc((void**)&out, sizeof(int)*sz) == hipSuccess;
	}

	__host__ void initialize(int size) {
		// launch kernel
		hipDeviceSynchronize();
		inSize = size;
		outSize = 0;
	}

public:
	__host__ Worklist(int npoints, int nedges) {
		if(!allocate(nedges)) {
			fprintf(stderr, "ERROR: %s\n", "Memory allocation error @Worklist");
		}
		initialize(npoints);
		insize = npoints
	}

	__device__ void push(int val) {
		int i = atomicAdd(&outSize, 1);
		out[i] = val;
	}

	__device__ int get(int pos) {
		return in[pos];
	}

	__device__ int get() {
		int i = atomicAdd(&inSize, 1);
		return in[i];
	}

	__host__ void swap() {
		// launch compute kernel
		inSize = outSize;
		outSize = 0;
		int *temp = in;
		in = out;
		out = temp;
	}

	~Worklist() {
	#ifndef __CUDA_ARCH__
		hipFree(in);
		hipFree(out);
	#endif
	}
};